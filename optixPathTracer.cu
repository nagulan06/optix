#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include "optixPathTracer.h"
#include "random.h"

#include <sutil/vec_math.h>
#include <cuda/helpers.h>

#define EPS     1.19209290E-07F
#define TWO_PI  6.28318530717959f       //2*pi

const unsigned int WIDTH = 556;
const unsigned int HEIGHT = 549;
const unsigned int DEPTH = 560;

extern "C" {
    __constant__ Params params;
}



//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

struct RadiancePRD
{
    // TODO: move some state directly into payload registers?
    float3       emitted;
    float3       radiance;
    float3       attenuation;
    float3       origin;
    float3       direction;
    unsigned int seed;
    int          countEmitted;
    int          done;
    float        slen;
    float        dist_so_far;
    unsigned int mc_seed[4];
    //int          pad;
};


struct Onb
{
    __forceinline__ __device__ Onb(const float3& normal)
    {
        m_normal = normal;

        if (fabs(m_normal.x) > fabs(m_normal.z))
        {
            m_binormal.x = -m_normal.y;
            m_binormal.y = m_normal.x;
            m_binormal.z = 0;
        }
        else
        {
            m_binormal.x = 0;
            m_binormal.y = -m_normal.z;
            m_binormal.z = m_normal.y;
        }

        m_binormal = normalize(m_binormal);
        m_tangent = cross(m_binormal, m_normal);
    }

    __forceinline__ __device__ void inverse_transform(float3& p) const
    {
        p = p.x * m_tangent + p.y * m_binormal + p.z * m_normal;
    }

    float3 m_tangent;
    float3 m_binormal;
    float3 m_normal;
};


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------



static __forceinline__ __device__ float xorshift128p_nextf(unsigned long t[2]) {
    union {
        unsigned long  i;
        float f[2];
        unsigned int  u[2];
    } s1;
    const unsigned long s0 = t[1];
    s1.i = t[0];
    t[0] = s0;
    s1.i ^= s1.i << 23; // a
    t[1] = s1.i ^ s0 ^ (s1.i >> 18) ^ (s0 >> 5); // b, c
    s1.i = t[1] + s0;
    s1.u[0] = 0x3F800000U | (s1.u[0] >> 9);

    return s1.f[0] - 1.0f;
}

static __forceinline__ __device__ float mc_next_scatter(float g, unsigned long ran[2], float3* dir) {

    float nextslen;
    float sphi, cphi, tmp0, theta, stheta, ctheta, tmp1;
    float3 p;

    //random scattering length (normalized)
    nextslen = -log(xorshift128p_nextf(ran) + EPS);

    tmp0 = TWO_PI * xorshift128p_nextf(ran); //next arimuth angle
    sphi = sin(tmp0);
    cphi = cos(tmp0);

    if (g > EPS) {  //if g is too small, the distribution of theta is bad
        tmp0 = (1.f - g * g) / (1.f - g + 2.f * g * xorshift128p_nextf(ran));
        tmp0 *= tmp0;
        tmp0 = (1.f + g * g - tmp0) / (2.f * g);
        tmp0 = clamp(tmp0, -1.f, 1.f);

        theta = acos(tmp0);
        stheta = sqrt(1.f - tmp0 * tmp0);
        //stheta=MCX_MATHFUN(sin)(theta);
        ctheta = tmp0;
    }
    else {
        theta = acos(2.f * xorshift128p_nextf(ran) - 1.f);
        stheta = sin(theta);
        ctheta = cos(theta);
    }

    if (dir->z > -1.f + EPS && dir->z < 1.f - EPS) {
        tmp0 = 1.f - dir->z * dir->z;   //reuse tmp to minimize registers
        tmp1 = 1 / sqrt(tmp0);
        tmp1 = stheta * tmp1;

        p.x = tmp1 * (dir->x * dir->z * cphi - dir->y * sphi) + dir->x * ctheta;
        p.y = tmp1 * (dir->y * dir->z * cphi + dir->x * sphi) + dir->y * ctheta;
        p.z = -tmp1 * tmp0 * cphi + dir->z * ctheta;
    }
    else {
        p.x = stheta * cphi;
        p.y = stheta * sphi;
        p.z = (dir->z > 0.f) ? ctheta : -ctheta;
    }

    dir->x = p.x;
    dir->y = p.y;
    dir->z = p.z;
    return nextslen;
}


static __forceinline__ __device__ void* unpackPointer(unsigned int i0, unsigned int i1)
{
    const unsigned long long uptr = static_cast<unsigned long long>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}


static __forceinline__ __device__ void  packPointer(void* ptr, unsigned int& i0, unsigned int& i1)
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}


static __forceinline__ __device__ RadiancePRD* getPRD()
{
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>(unpackPointer(u0, u1));
}


static __forceinline__ __device__ void setPayloadOcclusion(bool occluded)
{
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}


static __forceinline__ __device__ void cosine_sample_hemisphere(const float u1, const float u2, float3& p)
{
    // Uniformly sample disk.
    const float r = sqrtf(u1);
    const float phi = 2.0f * M_PIf * u2;
    p.x = r * cosf(phi);
    p.y = r * sinf(phi);

    // Project up to hemisphere.
    p.z = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.y * p.y));
}


static __forceinline__ __device__ void traceRadiance(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    RadiancePRD* prd
)
{
    // TODO: deduce stride from num ray-types passed in params

    unsigned int u0, u1;
    packPointer(prd, u0, u1);
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,        // SBT offset
        RAY_TYPE_COUNT,           // SBT stride
        RAY_TYPE_RADIANCE,        // missSBTIndex
        u0, u1);
}


static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax
)
{
    unsigned int occluded = 0u;
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                    // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,      // SBT offset
        RAY_TYPE_COUNT,          // SBT stride
        RAY_TYPE_OCCLUSION,      // missSBTIndex
        occluded);
    return occluded;
}


//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__rg()
{
    const int    w = params.width;
    const int    h = params.height;
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V;
    const float3 W = params.W;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;

    unsigned int seed = tea<4>(idx.y * w + idx.x, subframe_index);
    unsigned int seed1 = tea<4>((idx.y * w + idx.x) + 1, subframe_index);
    unsigned int seed2 = tea<4>((idx.y * w + idx.x) + 2, subframe_index);
    unsigned int seed3 = tea<4>((idx.y * w + idx.x) + 3, subframe_index);

    float3 result = make_float3(0.0f);
    int i = params.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2(rnd(seed), rnd(seed));

        const float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(w),
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(h)
        ) - 1.0f;
        float3 ray_direction = normalize(d.x * U + d.y * V + W);
        float3 ray_origin = eye;

        RadiancePRD prd;
        prd.emitted = make_float3(0.f);
        prd.radiance = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.mc_seed[0] = seed;
        prd.mc_seed[1] = seed1;
        prd.mc_seed[2] = seed2;
        prd.mc_seed[3] = seed3;

        prd.slen = rnd(seed) * 10;
        prd.dist_so_far = 0.0f;

        int depth = 0;
        for (;; )
        {
            traceRadiance(
                params.handle,
                ray_origin,
                ray_direction,
                0.01f,  // tmin       // TODO: smarter offset
                1e16f,  // tmax
                &prd);

            result += prd.emitted;
            result += prd.radiance * prd.attenuation;

            if (prd.done || depth >= 5) // TODO RR, variable for depth
                break;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ++depth;
        }
    } while (--i);

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;
    float3         accum_color = result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    params.frame_buffer[image_index] = make_color(accum_color);
}


extern "C" __global__ void __miss__radiance()
{
    MissData* rt_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    RadiancePRD* prd = getPRD();

    prd->radiance = make_float3(rt_data->bg_color);
    prd->done = true;
}


extern "C" __global__ void __closesthit__occlusion()
{
    setPayloadOcclusion(true);
}


extern "C" __global__ void __closesthit__radiance()
{
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx = optixGetPrimitiveIndex();
    const float3 ray_dir = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx * 3;

    const float3 v0 = make_float3(rt_data->vertices[vert_idx_offset + 0]);
    const float3 v1 = make_float3(rt_data->vertices[vert_idx_offset + 1]);
    const float3 v2 = make_float3(rt_data->vertices[vert_idx_offset + 2]);
    const float3 N_0 = normalize(cross(v1 - v0, v2 - v0));

    const float3 N = faceforward(N_0, -ray_dir, N_0);

    const float dist_travelled = optixGetRayTmax();
    const float3 inters_point = optixGetWorldRayOrigin() + dist_travelled * ray_dir;

    RadiancePRD* prd = getPRD();

    if (prd->countEmitted)
        prd->emitted = rt_data->emission_color;
    else
        prd->emitted = make_float3(0.0f);


    // Compute the ray attenuation
 /*
    float distance2 = (prd->origin.x - inters_point.x) * (prd->origin.x - inters_point.x) + (prd->origin.y - inters_point.y) * (prd->origin.y - inters_point.y) + (prd->origin.z - inters_point.z) * (prd->origin.z - inters_point.z);
    float distance = sqrt(distance2);

    uint3 prev_index;
    for (int i = 0; i < distance; i++)
    {
        float3 curr_location = prd->origin + i * prd->direction;
        uint3 index = make_uint3(curr_location.x, curr_location.y, curr_location.z);
        if (i > 0 && prev_index == index)
            continue;
        prev_index = index;

        params.attenuation_buffer[index.x + (index.y + index.z * DEPTH) * WIDTH];
    }
*/

    unsigned int seed = prd->seed;

    // Ray has travelled past its scattering length
    if (prd->dist_so_far >= prd->slen)
    {
        prd->origin = prd->origin + prd->slen;

        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere(z1, z2, w_in);
        Onb onb(N);
        onb.inverse_transform(w_in);
        //prd->direction = w_in;
        //prd->slen = rnd(prd->seed) * 10;
        unsigned long rand[2];
        rand[0] = (unsigned long)prd->mc_seed[0] << 32 | prd->mc_seed[1];
        rand[1] = (unsigned long)prd->mc_seed[2] << 32 | prd->mc_seed[3];

        // value of g?
        prd->slen = mc_next_scatter(0, rand, &prd->direction);

    }
    // Ray has not reached scatter length
    else
    {
        prd->origin = inters_point;
        prd->dist_so_far += dist_travelled;
    }

    {
        prd->attenuation *= rt_data->diffuse_color;
        prd->countEmitted = false;
    }

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd->seed = seed;

    ParallelogramLight light = params.light;
    const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - inters_point);
    const float3 L = normalize(light_pos - inters_point);
    const float  nDl = dot(N, L);
    const float  LnDl = -dot(light.normal, L);

    float weight = 0.0f;
    if (nDl > 0.0f && LnDl > 0.0f)
    {
        {
            const float A = length(cross(light.v1, light.v2));
            weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
        }
    }

    prd->radiance += light.emission * weight;
}


